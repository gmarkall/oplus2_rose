#include "hip/hip_runtime.h"
#include "user_defined_types.h"
#include "op_datatypes.h"
#include "kernels.h"
__device__
#include <res_calc.h>
__global__

void op_cuda_res_calc(float *ind_arg0,int *ind_arg0_ptrs,int *ind_arg0_sizes,int *ind_arg0_offset,float *ind_arg1,int *ind_arg1_ptrs,int *ind_arg1_sizes,int *ind_arg1_offset,float *ind_arg2,int *ind_arg2_ptrs,int *ind_arg2_sizes,int *ind_arg2_offset,float *ind_arg3,int *ind_arg3_ptrs,int *ind_arg3_sizes,int *ind_arg3_offset,int *arg0_ptrs,int *arg1_ptrs,int *arg2_ptrs,int *arg3_ptrs,int *arg4_ptrs,int *arg5_ptrs,int *arg6_ptrs,int *arg7_ptrs,int *arg8_d,int block_offset,int *blkmap,int *offset,int *nelems,int *ncolors,int *colors)
{
  float arg6_l[4];
  float arg7_l[4];
  int arg8_l[1];
  extern __shared__ 
  char shared[];
  __shared__ 
  int *ind_arg0_ptr;
  __shared__ 
  int *ind_arg1_ptr;
  __shared__ 
  int *ind_arg2_ptr;
  __shared__ 
  int *ind_arg3_ptr;
  __shared__ 
  int ind_arg0_size;
  __shared__ 
  int ind_arg1_size;
  __shared__ 
  int ind_arg2_size;
  __shared__ 
  int ind_arg3_size;
  __shared__ 
  float *ind_arg0_s;
  __shared__ 
  float *ind_arg1_s;
  __shared__ 
  float *ind_arg2_s;
  __shared__ 
  float *ind_arg3_s;
  __shared__ 
  int *arg0_ptr;
  __shared__ 
  int *arg1_ptr;
  __shared__ 
  int *arg2_ptr;
  __shared__ 
  int *arg3_ptr;
  __shared__ 
  int *arg4_ptr;
  __shared__ 
  int *arg5_ptr;
  __shared__ 
  int *arg6_ptr;
  __shared__ 
  int *arg7_ptr;
  __shared__ 
  int *arg8;
  __shared__ 
  int nelem2;
  __shared__ 
  int ncolor;
  __shared__ 
  int *color;
  __shared__ 
  int blockId;
  __shared__ 
  int nelem;
  if (threadIdx.x == 0) {
    blockId = blkmap[blockIdx.x + block_offset];
    nelem = nelems[blockId];
    ncolor = ncolors[blockId];
    int cur_offset = offset[blockId];
    color = colors + cur_offset;
    nelem2 = blockDim.x * (1 + (nelem - 1) / blockDim.x);
    ind_arg0_size = ind_arg0_sizes[blockId];
    ind_arg1_size = ind_arg1_sizes[blockId];
    ind_arg2_size = ind_arg2_sizes[blockId];
    ind_arg3_size = ind_arg3_sizes[blockId];
    ind_arg0_ptr = ind_arg0_ptrs + ind_arg0_offset[blockId];
    ind_arg1_ptr = ind_arg1_ptrs + ind_arg1_offset[blockId];
    ind_arg2_ptr = ind_arg2_ptrs + ind_arg2_offset[blockId];
    ind_arg3_ptr = ind_arg3_ptrs + ind_arg3_offset[blockId];
    arg0_ptr = arg0_ptrs + cur_offset;
    arg1_ptr = arg1_ptrs + cur_offset;
    arg2_ptr = arg2_ptrs + cur_offset;
    arg3_ptr = arg3_ptrs + cur_offset;
    arg4_ptr = arg4_ptrs + cur_offset;
    arg5_ptr = arg5_ptrs + cur_offset;
    arg6_ptr = arg6_ptrs + cur_offset;
    arg7_ptr = arg7_ptrs + cur_offset;
    arg8 = arg8_d + cur_offset * 1;
    int nbytes = 0;
    ind_arg0_s = ((float *)(&shared[nbytes]));
    nbytes += ROUND_UP(ind_arg0_size * (sizeof(float ) * 2));
    ind_arg1_s = ((float *)(&shared[nbytes]));
    nbytes += ROUND_UP(ind_arg1_size * (sizeof(float ) * 4));
    ind_arg2_s = ((float *)(&shared[nbytes]));
    nbytes += ROUND_UP(ind_arg2_size * (sizeof(float ) * 1));
    ind_arg3_s = ((float *)(&shared[nbytes]));
  }
  __syncthreads();
  for (int n = threadIdx.x; n < ind_arg0_size; n += blockDim.x) {
    int ind_index = ind_arg0_ptr[n];
    ind_arg0_s[0+n*2] = ind_arg0[0+ind_index*2];
    ind_arg0_s[1+n*2] = ind_arg0[1+ind_index*2];
  }
  for (int n = threadIdx.x; n < ind_arg1_size; n += blockDim.x) {
    int ind_index = ind_arg1_ptr[n];
    ind_arg1_s[0+n*4] = ind_arg1[0+ind_index*4];
    ind_arg1_s[1+n*4] = ind_arg1[1+ind_index*4];
    ind_arg1_s[2+n*4] = ind_arg1[2+ind_index*4];
    ind_arg1_s[3+n*4] = ind_arg1[3+ind_index*4];
  }
  for (int n = threadIdx.x; n < ind_arg2_size; n += blockDim.x) {
    ind_arg2_s[n*1] = ind_arg2[ind_arg2_ptr[n]*1];
  }
  for (int n = threadIdx.x; n < ind_arg3_size; n += blockDim.x) {
    ind_arg3_s[0+n*4] = 0;
    ind_arg3_s[1+n*4] = 0;
    ind_arg3_s[2+n*4] = 0;
    ind_arg3_s[3+n*4] = 0;
  }
  __syncthreads();
  for (int n = threadIdx.x; n < nelem2; n += blockDim.x) {
    int col2 = -1;
    if (n < nelem) {
      arg6_l[0] = 0;
      arg6_l[1] = 0;
      arg6_l[2] = 0;
      arg6_l[3] = 0;
      arg7_l[0] = 0;
      arg7_l[1] = 0;
      arg7_l[2] = 0;
      arg7_l[3] = 0;
      arg8_l[0] =  *(arg8 + (n * 1 + 0));
      res_calc(ind_arg0_s + arg0_ptr[n] * 2,ind_arg0_s + arg1_ptr[n] * 2,ind_arg1_s + arg2_ptr[n] * 4,ind_arg1_s + arg3_ptr[n] * 4,ind_arg2_s + arg4_ptr[n] * 1,ind_arg2_s + arg5_ptr[n] * 1,arg6_l,arg7_l,arg8_l);
      col2 = color[n];
    }
    for (int col = 0; col < ncolor; ++col) {
      if (col == col2) {
        int ind_index = arg6_ptr[n];
        ind_arg3_s[0+ind_index*4] += arg6_l[0];
        ind_arg3_s[1+ind_index*4] += arg6_l[1];
        ind_arg3_s[2+ind_index*4] += arg6_l[2];
        ind_arg3_s[3+ind_index*4] += arg6_l[3];
        ind_index = arg7_ptr[n];
        ind_arg3_s[0+ind_index*4] += arg7_l[0];
        ind_arg3_s[1+ind_index*4] += arg7_l[1];
        ind_arg3_s[2+ind_index*4] += arg7_l[2];
        ind_arg3_s[3+ind_index*4] += arg7_l[3];
      }
      __syncthreads();
    }
  }
  for (int n = threadIdx.x; n < ind_arg3_size; n += blockDim.x) {
    int ind_index = ind_arg3_ptr[n];
    ind_arg3[0+ind_index*4] += ind_arg3_s[0+n*4];
    ind_arg3[1+ind_index*4] += ind_arg3_s[1+n*4];
    ind_arg3[2+ind_index*4] += ind_arg3_s[2+n*4];
    ind_arg3[3+ind_index*4] += ind_arg3_s[3+n*4];
  }
}


float op_par_loop_res_calc(const char *name,op_set set,struct op_dat<void> *arg0,int idx0,op_ptr *ptr0,enum op_access acc0,struct op_dat<void> *arg1,int idx1,op_ptr *ptr1,enum op_access acc1,struct op_dat<void> *arg2,int idx2,op_ptr *ptr2,enum op_access acc2,struct op_dat<void> *arg3,int idx3,op_ptr *ptr3,enum op_access acc3,struct op_dat<void> *arg4,int idx4,op_ptr *ptr4,enum op_access acc4,struct op_dat<void> *arg5,int idx5,op_ptr *ptr5,enum op_access acc5,struct op_dat<void> *arg6,int idx6,op_ptr *ptr6,enum op_access acc6,struct op_dat<void> *arg7,int idx7,op_ptr *ptr7,enum op_access acc7,struct op_dat<void> *arg8,int idx8,op_ptr *ptr8,enum op_access acc8)
{
  int nargs = 9;
  int ninds = 4;
  int gridsize = (set.size - 1) / BSIZE + 1;
  struct op_dat<void> args[9] = { *arg0,  *arg1,  *arg2,  *arg3,  *arg4,  *arg5,  *arg6,  *arg7,  *arg8};
  int idxs[9] = {idx0, idx1, idx2, idx3, idx4, idx5, idx6, idx7, -1};
  op_ptr ptrs[9] = { *ptr0,  *ptr1,  *ptr2,  *ptr3,  *ptr4,  *ptr5,  *ptr6,  *ptr7, OP_ID};
  int dims[9] = {arg0->dim, arg1->dim, arg2->dim, arg3->dim, arg4->dim, arg5->dim, arg6->dim, arg7->dim, arg8->dim};
  enum op_access accs[9] = {acc0, acc1, acc2, acc3, acc4, acc5, acc6, acc7, acc8};
  int inds[9] = {0, 0, 1, 1, 2, 2, 3, 3, -1};
  op_plan *Plan = plan(name,set,nargs,args,idxs,ptrs,dims,accs,ninds,inds);
  int block_offset = 0;
  int reduct_bytes = 0;
  int reduct_size = 0;
  int reduct_shared = reduct_size * (BSIZE / 2);
  int const_bytes = 0;
  float total_time = 0.00000F;
  for (int col = 0; col < Plan->ncolors; ++col) {
    int nblocks = Plan->ncolblk[col];
    int nshared = Plan->nshared;
hipEvent_t start, stop;
    float elapsed_time_ms = 0.00000F;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    op_cuda_res_calc<<<nblocks,BSIZE,nshared>>>(((float *)arg0->dat_d),Plan->ind_ptrs[0],Plan->ind_sizes[0],Plan->ind_offs[0],((float *)arg2->dat_d),Plan->ind_ptrs[1],Plan->ind_sizes[1],Plan->ind_offs[1],((float *)arg4->dat_d),Plan->ind_ptrs[2],Plan->ind_sizes[2],Plan->ind_offs[2],((float *)arg6->dat_d),Plan->ind_ptrs[3],Plan->ind_sizes[3],Plan->ind_offs[3],Plan->ptrs[0],Plan->ptrs[1],Plan->ptrs[2],Plan->ptrs[3],Plan->ptrs[4],Plan->ptrs[5],Plan->ptrs[6],Plan->ptrs[7],((int *)arg8->dat_d),block_offset,Plan->blkmap,Plan->offset,Plan->nelems,Plan->nthrcol,Plan->thrcol);
    hipEventRecord(stop,0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    total_time += elapsed_time_ms;
    hipDeviceSynchronize();
    block_offset += nblocks;
  }
  return total_time;
}

