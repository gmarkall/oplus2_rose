#include "hip/hip_runtime.h"
/*
  Open source copyright declaration based on BSD open source template:
  http://www.opensource.org/licenses/bsd-license.php

* Copyright (c) 2009, Mike Giles
* All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions are met:
*     * Redistributions of source code must retain the above copyright
*       notice, this list of conditions and the following disclaimer.
*     * Redistributions in binary form must reproduce the above copyright
*       notice, this list of conditions and the following disclaimer in the
*       documentation and/or other materials provided with the distribution.
*     * The name of Mike Giles may not be used to endorse or promote products
*       derived from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY Mike Giles ''AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
* WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
* DISCLAIMED. IN NO EVENT SHALL Mike Giles BE LIABLE FOR ANY
* DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
* (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
* LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
* ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
* SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

//
// header files
//

#include <stdlib.h>                                                         
#include <stdio.h>                                                          
#include <string.h>                                                         
#include <math.h>                                                           

#include <cutil_inline.h>
#include <hip/hip_math_constants.h>
#include "user_defined_types.h"
#include "op_datatypes.h"
#include <vector>

using namespace std;

//
// global variables
//

int OP_set_index=0,
    OP_map_index=0,
    OP_dat_index=0,
    OP_nplans   =0;

op_set         * OP_set_list[10];
op_map         * OP_map_list[10];
op_dat<void>   * OP_dat_list[10];
op_plan            OP_plans[100];

// arrays for global constants and reductions

int   OP_consts_bytes=0,    OP_reduct_bytes=0;
char *OP_consts_h, *OP_consts_d, *OP_reduct_h, *OP_reduct_d;

//
// OP functions
//

void op_init(int argc, char **argv){
  cutilDeviceInit(argc, argv);
}

void op_decl_const_i(const char* dat, int size, char const *name)
{
  // printf(" op_decl_const: name = %s, size = %d\n",name,sizeof(T)*dim);
  cutilSafeCall( hipMemcpyToSymbol(HIP_SYMBOL(name), dat, size) );
}

void fixup_op_set(op_set* set)
{
  set->index = OP_set_index;
  
  // Add to the global set list
  OP_set_list[OP_set_index++] = set;
}

void fixup_op_map(op_map* map)
{
  map->index = OP_map_index;
 
  // Add to the global map list
  OP_map_list[OP_map_index++] = map;
}

void fixup_op_dat_data(op_dat<void>* data)
{
  data->index = OP_dat_index;
  
  // Add to the global dat list
  OP_dat_list[OP_dat_index++] = data;

  cutilSafeCall(hipMalloc((void **)&(data->dat_d), data->size*data->set.size));
  cutilSafeCall(hipMemcpy(data->dat_d, data->dat, data->size*data->set.size, hipMemcpyHostToDevice));
}

void push_op_dat_as_const(op_dat<void>& data, int offset_bytes)
{
  data.dat   = OP_consts_h + offset_bytes;                                                  
  data.dat_d = OP_consts_d + offset_bytes;
  memcpy(data.dat, data.dat_t, data.size);
}

void push_op_dat_as_reduct(op_dat<void>& data, int offset_bytes)
{
  data.dat   = OP_reduct_h + offset_bytes;
  data.dat_d = OP_reduct_d + offset_bytes;
  memcpy(data.dat, data.dat_t, data.size);
}

void pop_op_dat_as_reduct(op_dat<void>& data)
{
  memcpy(data.dat_t, data.dat, data.size);
}



void op_diagnostic_output(){
  if (OP_DIAGS > 1) {
    printf("\n  OP diagnostic output\n");
    printf(  "  --------------------\n");

    printf("\n       set       size\n");
    printf(  "  -------------------\n");
    for(int n=0; n<OP_set_index; n++) {
      op_set set=*OP_set_list[n];
      printf("%10s %10d\n",set.name,set.size);
    }

    printf("\n       map        dim       from         to\n");
    printf(  "  -----------------------------------------\n");
    for(int n=0; n<OP_map_index; n++) {
      op_map map=*OP_map_list[n];
      printf("%10s %10d %10s %10s\n",map.name,map.dim,map.from.name,map.to.name);
    }

    printf("\n       dat        dim        set\n");
    printf(  "  ------------------------------\n");
    for(int n=0; n<OP_dat_index; n++) {
      op_dat<void> dat=*OP_dat_list[n];
      printf("%10s %10d %10s\n",dat.name,dat.dim,dat.set.name);
    }
    printf("\n");
  }
}

void op_exit(){
}


//
// comparison function for integer quicksort
//

int compare(const void *a2, const void *b2) {
  int *a = (int *)a2;
  int *b = (int *)b2;

  if (*a == *b)
    return 0;
  else
    if (*a < *b)
      return -1;
    else
      return 1;
}

//
// utility routine to move arrays to GPU device
//

#ifndef OP_x86

template <class T>
void mvHostToDevice(T **ptr, int size) {
  T *tmp;
  cutilSafeCall(hipMalloc((void **)&tmp, size));
  cutilSafeCall(hipMemcpy(tmp, *ptr, size, hipMemcpyHostToDevice));
  free(*ptr);
  *ptr = tmp;
}


//
// utility routine to copy dataset back to host
//
void op_fetch_data_i(op_dat<void> *data) {
  cutilSafeCall(hipMemcpy(data->dat, data->dat_d, data->size*data->set.size, hipMemcpyDeviceToHost));
  cutilSafeCall(hipDeviceSynchronize());
}


//
// utility routines to resize constant/reduct arrays, if necessary
//

void reallocConstArrays(int consts_bytes) {
  if (OP_consts_bytes>0) {
    free(OP_consts_h);
    cutilSafeCall(hipFree(OP_consts_d));
  }
  OP_consts_bytes = 4*consts_bytes;
  OP_consts_h = (char *) malloc(OP_consts_bytes);
  cutilSafeCall(hipMalloc((void **)&OP_consts_d, OP_consts_bytes));
}

void reallocReductArrays(int reduct_bytes) {
  if (OP_reduct_bytes>0) {
    free(OP_reduct_h);
    cutilSafeCall(hipFree(OP_reduct_d));
  }
  OP_reduct_bytes = 4*reduct_bytes;
  OP_reduct_h = (char *) malloc(OP_reduct_bytes);
  cutilSafeCall(hipMalloc((void **)&OP_reduct_d, OP_reduct_bytes));
}

//
// utility routine to move constant/reduct arrays
//

void mvConstArraysToDevice(int consts_bytes) {
  cutilSafeCall(hipMemcpy(OP_consts_d, OP_consts_h, consts_bytes,
                hipMemcpyHostToDevice));
}

void mvReductArraysToDevice(int reduct_bytes) {
  cutilSafeCall(hipMemcpy(OP_reduct_d, OP_reduct_h, reduct_bytes,
                hipMemcpyHostToDevice));
}

void mvReductArraysToHost(int reduct_bytes) {
  cutilSafeCall(hipMemcpy(OP_reduct_h, OP_reduct_d, reduct_bytes,
                hipMemcpyDeviceToHost));
}


//
// reduction routine for arbitrary datatypes
//
__device__ int OP_reduct_lock=0;  // important: must be initialised to 0

template < op_access reduction, class T >
__inline__ __device__ void op_reduction(volatile T *dat_g, T dat_l)
{
  int tid = threadIdx.x;
  int d   = blockDim.x>>1; 
  extern __shared__ T temp[];

  if (tid>=d) temp[tid-d] = dat_l;
  __syncthreads();

  if (tid<d) {
    switch (reduction) {
    case OP_INC:
      temp[tid] = temp[tid] + dat_l;
      break;
    case OP_MIN:
      if(dat_l<temp[tid]) temp[tid] = dat_l;
      break;
    case OP_MAX:
      if(dat_l>temp[tid]) temp[tid] = dat_l;
      break;
    }
  }

  for (d>>=1; d>warpSize; d>>=1) {
    __syncthreads();
    if (tid<d) {
      switch (reduction) {
      case OP_INC:
        temp[tid] = temp[tid] + temp[tid+d];
        break;
      case OP_MIN:
        if(temp[tid+d]<temp[tid]) temp[tid] = temp[tid+d];
        break;
      case OP_MAX:
        if(temp[tid+d]>temp[tid]) temp[tid] = temp[tid+d];
        break;
      }
    }
  }

  __syncthreads();

  if (tid<warpSize)
    for (; d>0; d>>=1)
      switch (reduction) {
      case OP_INC:
        temp[tid] = temp[tid] + temp[tid+d];
        break;
      case OP_MIN:
        if(temp[tid+d]<temp[tid]) temp[tid] = temp[tid+d];
        break;
      case OP_MAX:
        if(temp[tid+d]>temp[tid]) temp[tid] = temp[tid+d];
        break;
      }

  if (tid==0) {
    do {} while(atomicCAS(&OP_reduct_lock,0,1));  // set lock

    switch (reduction) {
    case OP_INC:
      *dat_g = *dat_g + temp[0];
      break;
    case OP_MIN:
      if(temp[0]<*dat_g) *dat_g = temp[0];
      break;
    case OP_MAX:
      if(temp[0]>*dat_g) *dat_g = temp[0];
      break;
    }

    __threadfence();                // ensure *dat_g update complete
    OP_reduct_lock = 0;             // free lock
  }

  __syncthreads();  // important to finish one reduction before the next
}

template < op_access reduction >
__inline__ __device__ void op_reduction2_1(volatile float *dat_g, float dat_l, void* reduct)
{
  int tid = threadIdx.x;
  int d   = blockDim.x>>1; 
  extern __shared__ float temp[];

  if (tid>=d) temp[tid-d] = dat_l;
  __syncthreads();

  if (tid<d) {
    switch (reduction) {
    case OP_INC:
      temp[tid] = temp[tid] + dat_l;
      break;
    case OP_MIN:
      if(dat_l<temp[tid]) temp[tid] = dat_l;
      break;
    case OP_MAX:
      if(dat_l>temp[tid]) temp[tid] = dat_l;
      break;
    }
  }

  for (d>>=1; d>warpSize; d>>=1) {
    __syncthreads();
    if (tid<d) {
      switch (reduction) {
      case OP_INC:
        temp[tid] = temp[tid] + temp[tid+d];
        break;
      case OP_MIN:
        if(temp[tid+d]<temp[tid]) temp[tid] = temp[tid+d];
        break;
      case OP_MAX:
        if(temp[tid+d]>temp[tid]) temp[tid] = temp[tid+d];
        break;
      }
    }
  }

  __syncthreads();

  if (tid<warpSize)
    for (; d>0; d>>=1)
      switch (reduction) {
      case OP_INC:
        temp[tid] = temp[tid] + temp[tid+d];
        break;
      case OP_MIN:
        if(temp[tid+d]<temp[tid]) temp[tid] = temp[tid+d];
        break;
      case OP_MAX:
        if(temp[tid+d]>temp[tid]) temp[tid] = temp[tid+d];
        break;
      }

  if (tid==0) {
      float* block_reduct = (float*)reduct;
      block_reduct[blockIdx.x] = temp[0];
  }

  __syncthreads();  // important to finish one reduction before the next
}

template < op_access reduction >
__inline__ __device__ void op_reduction2_2(volatile float *dat_g, void* reduct, int gridDim)
{
	float* block_reduct = (float*)reduct;
	//if(blockIdx.x == 0 && threadIdx.x == 0)
	{
		for(int i=0; i<gridDim; i++)
		{
			switch (reduction) {
		  case OP_INC:
		    *dat_g = *dat_g + block_reduct[i];
		    break;
		  case OP_MIN:
		    if(block_reduct[i]<*dat_g) *dat_g = block_reduct[i];
		    break;
		  case OP_MAX:
		    if(block_reduct[i]>*dat_g) *dat_g = block_reduct[i];
     	 	break;
   		}
		}
	}
  __syncthreads();  // important to finish one reduction before the next
}

#endif

//
// declaration of plan check routine
//

void OP_plan_check(op_plan, int, int *,int);


//
// find existing execution plan, or construct a new one
//

extern op_plan * plan(char const * name, op_set set, int nargs, op_dat<void> *args, int *idxs,
      op_map *maps, int *dims, op_access *accs, int ninds, int *inds){

  // first look for an existing execution plan

  int ip=0, match=0;

  while (match==0 && ip<OP_nplans) {
    if ( (strcmp(name,        OP_plans[ip].name)==0)
             && (set.index == OP_plans[ip].set_index)
             && (nargs     == OP_plans[ip].nargs) ) {
      match = 1;
      for (int m=0; m<nargs; m++) {
        match = match && (args[m].index == OP_plans[ip].arg_idxs[m])
                      && (idxs[m]       == OP_plans[ip].idxs[m])
                      && (maps[m].index == OP_plans[ip].map_idxs[m])
                      && (dims[m]       == OP_plans[ip].dims[m])
                      && (accs[m]       == OP_plans[ip].accs[m]);
      }
    }
    ip++;
  }

  if (match) {
    ip--;
    if (OP_DIAGS > 1) printf(" old execution plan #%d\n",ip);
    return &(OP_plans[ip]);
  }
  else {
    if (OP_DIAGS > 1) printf(" new execution plan #%d\n",ip);
  }

  // consistency checks

  if (OP_DIAGS > 0) {
    for (int m=0; m<nargs; m++) {
      if (idxs[m] == -1) {
        //if (maps[m].index != -1) {
        if (maps[m].map != NULL) {
          printf("error2: wrong pointer for arg %d in kernel \"%s\"\n",m,name);
          printf("maps[m].index = %d\n",maps[m].index);
          printf("maps[m].name  = %s\n",maps[m].name);
          exit(1);
        }
      }
      else {
        if (set.index         != maps[m].from.index ||
            args[m].set.index != maps[m].to.index) {
          printf("error: wrong pointer for arg %d in kernel \"%s\"\n",m,name);
          exit(1);
        }
        if (maps[m].dim <= idxs[m]) {
          printf(" %d %d",maps[m].dim,idxs[m]);
          printf("error: invalid pointer index for arg %d in kernel \"%s\"\n",m,name);
          exit(1);
        }
      }
      if (args[m].dim != dims[m] && args[m].set.size>0) {
        printf("error: wrong dimension for arg %d in kernel \"%s\"\n",m,name);
        exit(1);
      }
    }
  }

  // set blocksize and number of blocks
  int bsize   = 256;   // blocksize
  int nblocks = (set.size-1)/bsize + 1;
	bool smartpartition = false;	

	if(set.partinfo != NULL)
	{
		smartpartition = true;
		nblocks = set.partinfo->size();
		for(int q=0; q<nblocks; q++)
		{
			bsize = MAX(set.partinfo->at(q), bsize);
		}
	}
	
  printf(" number of blocks = %d\n",nblocks);

  // allocate memory for new execution plan and store input arguments

  OP_plans[ip].arg_idxs  = (int *)malloc(nargs*sizeof(int));
  OP_plans[ip].idxs      = (int *)malloc(nargs*sizeof(int));
  OP_plans[ip].map_idxs  = (int *)malloc(nargs*sizeof(int));
  OP_plans[ip].dims      = (int *)malloc(nargs*sizeof(int));
  OP_plans[ip].accs      = (op_access *)malloc(nargs*sizeof(op_access));

  OP_plans[ip].nthrcol   = (int *)malloc(nblocks*sizeof(int));
  OP_plans[ip].thrcol    = (int *)calloc(set.size,sizeof(int));
  OP_plans[ip].offset    = (int *)malloc(nblocks*sizeof(int));
  OP_plans[ip].ind_maps  = (int **)malloc(ninds*sizeof(int *));
  OP_plans[ip].ind_offs  = (int **)malloc(ninds*sizeof(int *));
  OP_plans[ip].ind_sizes = (int **)malloc(ninds*sizeof(int *));
  OP_plans[ip].maps      = (int **)malloc(nargs*sizeof(int *));
  OP_plans[ip].nelems    = (int *)malloc(nblocks*sizeof(int));
  OP_plans[ip].ncolblk   = (int *)calloc(set.size,sizeof(int)); // max possibly needed
  OP_plans[ip].blkmap    = (int *)calloc(nblocks,sizeof(int));

  for (int m=0; m<ninds; m++) {
    int count = 0;
    for (int m2=0; m2<nargs; m2++)
      if (inds[m2]==m) count++;
    OP_plans[ip].ind_maps[m]  = (int *)malloc(count*set.size*sizeof(int));
    OP_plans[ip].ind_offs[m]  = (int *)malloc(nblocks*sizeof(int));
    OP_plans[ip].ind_sizes[m] = (int *)malloc(nblocks*sizeof(int));
  }

  for (int m=0; m<nargs; m++) {
    OP_plans[ip].maps[m]     = (int *)malloc(set.size*sizeof(int));

    OP_plans[ip].arg_idxs[m] = args[m].index;
    OP_plans[ip].idxs[m]     = idxs[m];
    OP_plans[ip].map_idxs[m] = maps[m].index;
    OP_plans[ip].dims[m]     = dims[m];
    OP_plans[ip].accs[m]     = accs[m];
  }

  OP_plans[ip].name      = name;
  OP_plans[ip].set_index = set.index;
  OP_plans[ip].nargs     = nargs;
    
  OP_nplans++;

  // allocate working arrays

  uint **work;
  work = (uint **)malloc(ninds*sizeof(uint *));

  for (int m=0; m<ninds; m++) {
    int m2 = 0;
    while(inds[m2]!=m) m2++;

    work[m] = (uint *)malloc(maps[m2].to.size*sizeof(uint));
  }

  int *work2;
  work2 = (int *)malloc(nargs*bsize*sizeof(int));  // max possibly needed

  // process set one block at a time

  int *nindirect;
  nindirect = (int *)calloc(ninds,sizeof(int));  // total number of indirect elements
	
	int bs_offset = 0;
  for (int b=0; b<nblocks; b++) {
		
    int  bs   = MIN(bsize, set.size - bs_offset);
		if(smartpartition) bs = set.partinfo->at(b);

    OP_plans[ip].offset[b] = bs_offset;    // offset for block
    OP_plans[ip].nelems[b] = bs;         			// size of block

    // loop over indirection sets

    for (int m=0; m<ninds; m++) {

      // build the list of elements indirectly referenced in this block

      int ne = 0;  // number of elements
      for (int m2=0; m2<nargs; m2++) {
        if (inds[m2]==m) {
          for (int e=bs_offset; e<bs_offset+bs; e++)
            work2[ne++] = maps[m2].map[idxs[m2]+e*maps[m2].dim];
				}
      }

      // sort them, then eliminate duplicates

      qsort(work2,ne,sizeof(int),compare);
        
      int e = 0;
      int p = 0;
      while (p<ne) {
        work2[e] = work2[p];
        while (p<ne && work2[p]==work2[e]) p++;
        e++;
      }
      ne = e;  // number of distinct elements

      /*
      if (OP_DIAGS > 5) {
        printf(" indirection set %d: ",m);
        for (int e=0; e<ne; e++) printf(" %d",work2[e]);
        printf(" \n");
      }
      */

      // store mapping and renumbered pointers in execution plan

      for (int e=0; e<ne; e++) {
        OP_plans[ip].ind_maps[m][nindirect[m]++] = work2[e];
        work[m][work2[e]] = e;   // inverse mapping
      }

      for (int m2=0; m2<nargs; m2++) {
        if (inds[m2]==m) {
          for (int e=bs_offset; e<bs_offset+bs; e++)
            OP_plans[ip].maps[m2][e] = work[m][maps[m2].map[idxs[m2]+e*maps[m2].dim]];
	}
      }

      if (b==0) {
        OP_plans[ip].ind_offs[m][b]  = 0;
        OP_plans[ip].ind_sizes[m][b] = nindirect[m];
      }
      else {
        OP_plans[ip].ind_offs[m][b]  = OP_plans[ip].ind_offs[m][b-1]
                                     + OP_plans[ip].ind_sizes[m][b-1];
        OP_plans[ip].ind_sizes[m][b] = nindirect[m] - OP_plans[ip].ind_offs[m][b];
      }
    }


    // print out re-numbered pointers

    /*
    for (int m=0; m<nargs; m++) {
      if (inds[m]>=0) {
        printf(" pointer table %d\n",m);
        for (int e=0; e<set.size; e++)
          printf(" map = %d\n",OP_plans[ip].maps[m][e]);
      }
    }

    for (int m=0; m<ninds; m++) {
      printf(" indirect set %d\n",m);
      for (int b=0; b<nblocks; b++) {
        printf("OP_plans[ip].ind_sizes[m][b] = %d\n", OP_plans[ip].ind_sizes[m][b]);
        printf("OP_plans[ip].ind_offs[m][b] = %d\n", OP_plans[ip].ind_offs[m][b]);
      }
    }
    */

    // now colour main set elements

    int repeat  = 1;
    int ncolor  = 0;
    int ncolors = 0;

    while (repeat) {
      repeat = 0;

      for (int m=0; m<nargs; m++) {
        if (inds[m]>=0)
          for (int e=bs_offset; e<bs_offset+bs; e++)
            work[inds[m]][maps[m].map[idxs[m]+e*maps[m].dim]] = 0;  // zero out color array
      }

      for (int e=bs_offset; e<bs_offset+bs; e++) {
        if (OP_plans[ip].thrcol[e]==0) {
          int mask = 0;
          for (int m=0; m<nargs; m++)
            if (inds[m]>=0 && accs[m]==OP_INC)
              mask |= work[inds[m]][maps[m].map[idxs[m]+e*maps[m].dim]]; // set bits of mask

          int color = ffs(~mask) - 1;   // find first bit not set
          if (color==-1) {              // run out of colors on this pass
            repeat = 1;
          }
          else {
            OP_plans[ip].thrcol[e] = ncolor+color;
            mask    = 1 << color;
            ncolors = MAX(ncolors, ncolor+color+1);

            for (int m=0; m<nargs; m++)
              if (inds[m]>=0 && accs[m]==OP_INC)
                work[inds[m]][maps[m].map[idxs[m]+e*maps[m].dim]] |= mask; // set color bit
          }
        }
      }

      ncolor += 32;   // increment base level
    }

    OP_plans[ip].nthrcol[b] = ncolors;  // number of thread colors in this block

    // reorder elements by color?
		

		// update bs_offset
		if(smartpartition)
			bs_offset += set.partinfo->at(b);
		else
			bs_offset += bsize;
  }


  // colour the blocks, after initialising colors to 0
	
  int *blk_col;
  blk_col = (int *)calloc(nblocks,sizeof(int));

  int repeat  = 1;
  int ncolor  = 0;
  int ncolors = 0;
	
  while (repeat) {
    repeat = 0;

    for (int m=0; m<nargs; m++) {
      if (inds[m]>=0) 
        for (int e=0; e<maps[m].to.size; e++)
          work[inds[m]][e] = 0;               // zero out color arrays
    }
		bs_offset = 0;
    for (int b=0; b<nblocks; b++) {
      if (blk_col[b] == 0) {          // color not yet assigned to block
        int  bs   = MIN(bsize, set.size - bs_offset);
				if(smartpartition) bs = set.partinfo->at(b);
        uint mask = 0;

        for (int m=0; m<nargs; m++) {
          if (inds[m]>=0) 
            for (int e=bs_offset; e<bs_offset+bs; e++)
              mask |= work[inds[m]][maps[m].map[idxs[m]+e*maps[m].dim]]; // set bits of mask
        }

        int color = ffs(~mask) - 1;   // find first bit not set
        if (color==-1) {              // run out of colors on this pass
          repeat = 1;
        }
        else {
          blk_col[b] = ncolor + color;
          mask    = 1 << color;
          ncolors = MAX(ncolors, ncolor+color+1);

          for (int m=0; m<nargs; m++) {
            if (inds[m]>=0)
              for (int e=bs_offset; e<bs_offset+bs; e++)
                work[inds[m]][maps[m].map[idxs[m]+e*maps[m].dim]] |= mask;
          }
        }
      }
			// update bs_offset
			if(smartpartition)
				bs_offset += set.partinfo->at(b);
			else
				bs_offset += bsize;
    }

    ncolor += 32;   // increment base level
  }


  // store block mapping and number of blocks per color


  OP_plans[ip].ncolors = ncolors;

  for (int b=0; b<nblocks; b++)
    OP_plans[ip].ncolblk[blk_col[b]]++;  // number of blocks of each color

  for (int c=1; c<ncolors; c++)
    OP_plans[ip].ncolblk[c] += OP_plans[ip].ncolblk[c-1]; // cumsum

  for (int c=0; c<ncolors; c++) work2[c]=0;

  for (int b=0; b<nblocks; b++) {
    int c  = blk_col[b];
    int b2 = work2[c];     // number of preceding blocks of this color
    if (c>0) b2 += OP_plans[ip].ncolblk[c-1];  // plus previous colors

    OP_plans[ip].blkmap[b2] = b;

    work2[c]++;            // increment counter
  }

  for (int c=ncolors-1; c>0; c--)
    OP_plans[ip].ncolblk[c] -= OP_plans[ip].ncolblk[c-1]; // undo cumsum

  // reorder blocks by color?


  // work out shared memory requirements

  OP_plans[ip].nshared = 0;

  for (int b=0; b<nblocks; b++) {
    int nbytes = 0;
    for (int m=0; m<ninds; m++) {
      int m2 = 0;
      while(inds[m2]!=m) m2++;

      nbytes += ROUND_UP(OP_plans[ip].ind_sizes[m][b]*args[m2].size);
    }
    OP_plans[ip].nshared = MAX(OP_plans[ip].nshared,nbytes);
  }

  // printf(" shared memory = %d bytes \n",OP_plans[ip].nshared);


  // validate plan info

  OP_plan_check(OP_plans[ip],ninds,inds,ncolors);


  // move plan arrays to GPU

  for (int m=0; m<ninds; m++) {
    mvHostToDevice(&(OP_plans[ip].ind_maps[m]), sizeof(int)*nindirect[m]);
    mvHostToDevice(&(OP_plans[ip].ind_sizes[m]),sizeof(int)*nblocks);
    mvHostToDevice(&(OP_plans[ip].ind_offs[m]), sizeof(int)*nblocks);
  }

  for (int m=0; m<nargs; m++) {
    if (inds[m]>=0)
      mvHostToDevice(&(OP_plans[ip].maps[m]), sizeof(int)*set.size);
  }

  mvHostToDevice(&(OP_plans[ip].nthrcol),sizeof(int)*nblocks);
  mvHostToDevice(&(OP_plans[ip].thrcol ),sizeof(int)*set.size);
  mvHostToDevice(&(OP_plans[ip].offset ),sizeof(int)*nblocks);
  mvHostToDevice(&(OP_plans[ip].nelems ),sizeof(int)*nblocks);
  mvHostToDevice(&(OP_plans[ip].blkmap ),sizeof(int)*nblocks);

  // free work arrays

  for (int m=0; m<ninds; m++) free(work[m]);
  free(work);
  free(work2);
  free(blk_col);
  free(nindirect);

  // return pointer to plan

  return &(OP_plans[ip]);
}


void OP_plan_check(op_plan OP_plan, int ninds, int *inds, int ncolors) {

  int err, ntot;

  op_set set = *OP_set_list[OP_plan.set_index];

  int nblock = 0;
  for (int col=0; col<OP_plan.ncolors; col++) nblock += OP_plan.ncolblk[col];

  //
  // check total size
  //

  int nelem = 0;
  for (int n=0; n<nblock; n++) nelem += OP_plan.nelems[n];

  if (nelem != set.size) {
    printf(" *** OP_plan_check: nelems error \n");
  }
  else {
    printf(" *** OP_plan_check: nelems   OK \n");
  }

  //
  // check offset and nelems are consistent
  //

  err  = 0;
  ntot = 0;

  for (int n=0; n<nblock; n++) {
    err  += (OP_plan.offset[n] != ntot);
    ntot +=  OP_plan.nelems[n];
  }

  if (err != 0) {
    printf(" *** OP_plan_check: offset error \n");
  }
  else {
    printf(" *** OP_plan_check: offset   OK \n");
  }

  //
  // check blkmap permutation
  //

  int *blkmap = (int *) malloc(nblock*sizeof(int));
  for (int n=0; n<nblock; n++) blkmap[n] = OP_plan.blkmap[n];
  qsort(blkmap,nblock,sizeof(int),compare);

  err = 0;
  for (int n=0; n<nblock; n++) err += (blkmap[n] != n);

  free(blkmap);

  if (err != 0) {
    printf(" *** OP_plan_check: blkmap error \n");
  }
  else {
    printf(" *** OP_plan_check: blkmap   OK \n");
  }

  //
  // check ind_offs and ind_sizes are consistent
  //

  err  = 0;

  for (int i = 0; i<ninds; i++) {
    ntot = 0;

    for (int n=0; n<nblock; n++) {
      err  += (OP_plan.ind_offs[i][n] != ntot);
      ntot +=  OP_plan.ind_sizes[i][n];
    }
  }

  if (err != 0) {
    printf(" *** OP_plan_check: ind_offs error \n");
  }
  else {
    printf(" *** OP_plan_check: ind_offs OK \n");
  }

  //
  // check ind_maps correctly ordered within each block
  // and indices within range
  //

  err = 0;

  for (int m = 0; m<ninds; m++) {
    int m2 = 0;
    while(inds[m2]!=m) m2++;
    int set_size = (*OP_map_list[OP_plan.map_idxs[m2]]).to.size;

    ntot = 0;

    for (int n=0; n<nblock; n++) {
      int last = -1;
      for (int e=ntot; e<ntot+OP_plan.ind_sizes[m][n]; e++) {
        err  += (OP_plan.ind_maps[m][e] <= last);
        last  = OP_plan.ind_maps[m][e]; 
      }
      err  += (last >= set_size);
      ntot +=  OP_plan.ind_sizes[m][n];
    }
  }

  if (err != 0) {
    printf(" *** OP_plan_check: ind_maps error \n");
  }
  else {
    printf(" *** OP_plan_check: ind_maps OK \n");
  }

  //
  // check maps (most likely source of errors)
  //

  err = 0;

  for (int m=0; m<OP_plan.nargs; m++) {
    if (OP_plan.map_idxs[m]>=0) {
      op_map map = *OP_map_list[OP_plan.map_idxs[m]];
      int    m2  = inds[m];

      ntot = 0;
      for (int n=0; n<nblock; n++) {
        for (int e=ntot; e<ntot+OP_plan.nelems[n]; e++) {
          int p_local  = OP_plan.maps[m][e];
          int p_global = OP_plan.ind_maps[m2][p_local+OP_plan.ind_offs[m2][n]];
          err += (p_global != map.map[OP_plan.idxs[m] + e * map.dim]);
        }
        ntot +=  OP_plan.nelems[n];
				//printf("\nPLAN: %d %d", n, OP_plan.nelems[n]);
      }
    }
  }

  if (err != 0) {
    printf(" *** OP_plan_check: maps error \n");
  }
  else {
    printf(" *** OP_plan_check: maps     OK \n");
  }


  //
  // check thread and block coloring
  //
  printf(" *** Number of Colors:%d \n", ncolors);

  return;
}
