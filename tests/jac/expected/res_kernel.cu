#include "hip/hip_runtime.h"
#include "user_defined_types.h"
#include "op_datatypes.h"
#include "kernels.h"
__device__
#include <res.h>
__global__

void op_cuda_res(float *ind_arg0,int *ind_arg0_ptrs,int *ind_arg0_sizes,int *ind_arg0_offset,float *ind_arg1,int *ind_arg1_ptrs,int *ind_arg1_sizes,int *ind_arg1_offset,double *arg0_d,int *arg1_ptrs,int *arg2_ptrs,float *arg3,int block_offset,int *blkmap,int *offset,int *nelems,int *ncolors,int *colors)
{
  double arg0_l[1];
  float arg2_l[1];
  extern __shared__ 
  char shared[];
  __shared__ 
  int *ind_arg0_ptr;
  __shared__ 
  int *ind_arg1_ptr;
  __shared__ 
  int ind_arg0_size;
  __shared__ 
  int ind_arg1_size;
  __shared__ 
  float *ind_arg0_s;
  __shared__ 
  float *ind_arg1_s;
  __shared__ 
  double *arg0;
  __shared__ 
  int *arg1_ptr;
  __shared__ 
  int *arg2_ptr;
  __shared__ 
  int nelem2;
  __shared__ 
  int ncolor;
  __shared__ 
  int *color;
  __shared__ 
  int blockId;
  __shared__ 
  int nelem;
  if (threadIdx.x == 0) {
    blockId = blkmap[blockIdx.x + block_offset];
    nelem = nelems[blockId];
    ncolor = ncolors[blockId];
    int cur_offset = offset[blockId];
    color = colors + cur_offset;
    nelem2 = blockDim.x * (1 + (nelem - 1) / blockDim.x);
    ind_arg0_size = ind_arg0_sizes[blockId];
    ind_arg1_size = ind_arg1_sizes[blockId];
    ind_arg0_ptr = ind_arg0_ptrs + ind_arg0_offset[blockId];
    ind_arg1_ptr = ind_arg1_ptrs + ind_arg1_offset[blockId];
    arg0 = arg0_d + cur_offset * 1;
    arg1_ptr = arg1_ptrs + cur_offset;
    arg2_ptr = arg2_ptrs + cur_offset;
    int nbytes = 0;
    ind_arg0_s = ((float *)(&shared[nbytes]));
    nbytes += ROUND_UP(ind_arg0_size * (sizeof(float ) * 1));
    ind_arg1_s = ((float *)(&shared[nbytes]));
  }
  __syncthreads();
  for (int n = threadIdx.x; n < ind_arg0_size; n += blockDim.x) {
    ind_arg0_s[n*1] = ind_arg0[ind_arg0_ptr[n]*1];
  }
  for (int n = threadIdx.x; n < ind_arg1_size; n += blockDim.x) {
    ind_arg1_s[n*1] = 0;
  }
  __syncthreads();
  for (int n = threadIdx.x; n < nelem2; n += blockDim.x) {
    int col2 = -1;
    if (n < nelem) {
      arg2_l[0] = 0;
      arg0_l[0] =  *(arg0 + (n * 1 + 0));
      res(arg0_l,ind_arg0_s + arg1_ptr[n] * 1,arg2_l,arg3);
      col2 = color[n];
    }
    for (int col = 0; col < ncolor; ++col) {
      if (col == col2) {
        ind_arg1_s[arg2_ptr[n]*1] += arg2_l[0];
      }
      __syncthreads();
    }
  }
  for (int n = threadIdx.x; n < ind_arg1_size; n += blockDim.x) {
    ind_arg1[ind_arg1_ptr[n]*1] = ind_arg1_s[n*1];
  }
}


float op_par_loop_res(const char *name,op_set set,struct op_dat<void> *arg0,int idx0,op_ptr *ptr0,enum op_access acc0,struct op_dat<void> *arg1,int idx1,op_ptr *ptr1,enum op_access acc1,struct op_dat<void> *arg2,int idx2,op_ptr *ptr2,enum op_access acc2,struct op_dat<void> *arg3,int idx3,op_ptr *ptr3,enum op_access acc3)
{
  int nargs = 4;
  int ninds = 2;
  int gridsize = (set.size - 1) / BSIZE + 1;
  struct op_dat<void> args[4] = { *arg0,  *arg1,  *arg2,  *arg3};
  int idxs[4] = {-1, idx1, idx2, -1};
  op_ptr ptrs[4] = {OP_ID,  *ptr1,  *ptr2, OP_ID};
  int dims[4] = {arg0->dim, arg1->dim, arg2->dim, arg3->dim};
  enum op_access accs[4] = {acc0, acc1, acc2, acc3};
  int inds[4] = {-1, 0, 1, -1};
  op_plan *Plan = plan(name,set,nargs,args,idxs,ptrs,dims,accs,ninds,inds);
  int block_offset = 0;
  int reduct_bytes = 0;
  int reduct_size = 0;
  int reduct_shared = reduct_size * (BSIZE / 2);
  int const_bytes = 0;
  const_bytes += ROUND_UP(1 * sizeof(float ));
  reallocConstArrays(const_bytes);
  const_bytes = 0;
  push_op_dat_as_const(*arg3,const_bytes);
  const_bytes += ROUND_UP(1 * sizeof(float ));
  mvConstArraysToDevice(const_bytes);
  float total_time = 0.00000F;
  for (int col = 0; col < Plan->ncolors; ++col) {
    int nblocks = Plan->ncolblk[col];
    int nshared = Plan->nshared;
hipEvent_t start, stop;
    float elapsed_time_ms = 0.00000F;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);
    op_cuda_res<<<nblocks,BSIZE,nshared>>>(((float *)arg1->dat_d),Plan->ind_ptrs[0],Plan->ind_sizes[0],Plan->ind_offs[0],((float *)arg2->dat_d),Plan->ind_ptrs[1],Plan->ind_sizes[1],Plan->ind_offs[1],((double *)arg0->dat_d),Plan->ptrs[1],Plan->ptrs[2],((float *)arg3->dat_d),block_offset,Plan->blkmap,Plan->offset,Plan->nelems,Plan->nthrcol,Plan->thrcol);
    hipEventRecord(stop,0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time_ms,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    total_time += elapsed_time_ms;
    hipDeviceSynchronize();
    block_offset += nblocks;
  }
  return total_time;
}

