#include "hip/hip_runtime.h"
#include "user_defined_types.h"
#include "op_datatypes.h"
#include "kernels.h"
__device__
#include <save_soln.h>
__global__

void op_cuda_save_soln(float *arg0,float *arg1,int set_size)
{
  for (int n = threadIdx.x + blockIdx.x * blockDim.x; n < set_size; n += blockDim.x * gridDim.x) {
    save_soln(arg0 + n * 4,arg1 + n * 4);
  }
}


float op_par_loop_save_soln(const char *name,op_set set,struct op_dat<void> *arg0,int idx0,op_map *map0,enum op_access acc0,struct op_dat<void> *arg1,int idx1,op_map *map1,enum op_access acc1)
{
  int bsize = OP_block_size;
  int gridsize = (set.size - 1) / bsize + 1;
  int reduct_bytes = 0;
  int reduct_size = 0;
  int reduct_shared = reduct_size * (OP_block_size / 2);
  int const_bytes = 0;
hipEvent_t start, stop;
  float elapsed_time_ms = 0.00000F;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  op_cuda_save_soln<<<gridsize,bsize,reduct_shared>>>(((float *)arg0->dat_d),((float *)arg1->dat_d),set.size);
  hipEventRecord(stop,0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&elapsed_time_ms,start,stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  return elapsed_time_ms;
}

